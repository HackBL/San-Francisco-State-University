#include "hip/hip_runtime.h"
/*
 *  Compile: nvcc findRedsGPU.cu -o output
 *
 *  Run: ./output
 *
 */



#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUMPARTICLES 32768
#define NEIGHBORHOOD .05
#define THREADSPERBLOCK 128

void initPos(float *);
float findDistance(float *, int, int);
__device__ float findDistanceGPU(float *, int, int);
void dumpResults(int index[]);

__global__ void findRedsGPU(float *p, int *numI);

int main() {
    hipEvent_t start, stop;
    float time;
    
    float *pos, *dpos;
    int *numReds, *dnumReds;
    
    pos = (float *) malloc(NUMPARTICLES * 4 * sizeof(float));
    numReds = (int *) malloc(NUMPARTICLES * sizeof(int));
    
    initPos(pos);
    
    // your code to allocate device arrays for pos and numReds go here
    hipMalloc((void **)&dpos, NUMPARTICLES * 4 * sizeof(float));
    hipMalloc((void **)&dnumReds, NUMPARTICLES * sizeof(int));
    
    // copy the host array pos[] to the corresponding device array
    hipMemcpy(dpos, pos, NUMPARTICLES * 4 * sizeof(float), hipMemcpyHostToDevice);
    
    // create timer events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    
    /* invoke kernel findRedsGPU here */
    findRedsGPU<<<NUMPARTICLES/THREADSPERBLOCK,THREADSPERBLOCK>>>(dpos,dnumReds);
    
    hipDeviceSynchronize();
    
    // your code to copy results to numReds[] go here
    hipMemcpy(numReds,dnumReds, NUMPARTICLES * sizeof(int), hipMemcpyDeviceToHost);
    
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    printf("Elapsed time = %f\n", time);
    
    dumpResults(numReds);
    
}

void initPos(float *p) {
    
    // your code for initializing pos goes here
    int i;
    int roll;
    for (i=0; i<NUMPARTICLES; i++) {
        p[i*4] = rand() / (float) RAND_MAX;
        p[i*4+1] = rand() / (float) RAND_MAX;
        p[i*4+2] = rand() / (float) RAND_MAX;
        roll = rand() % 3;
        if (roll == 0)
            p[i*4+3] = 0xff0000;
        else if (roll == 1)
            p[i*4+3] = 0x00ff00;
        else
            p[i*4+3] = 0x0000ff;
    }
}

__device__ float findDistanceGPU(float *p, int i, int j) {
    
    // your code for calculating distance for particle i and j
    float dx, dy, dz;
    
    dx = p[i*4] - p[j*4];
    dy = p[i*4+1] - p[j*4+1];
    dz = p[i*4+2] - p[j*4+2];
    
    return(sqrt(dx*dx + dy*dy + dz*dz));
}

__global__ void findRedsGPU(float *p, int *numI) {
    
    // your code for counting red particles goes here
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    float distance;
    numI[i] = 0;
    
    for (j = 0; j < NUMPARTICLES; j++) {
        if (i != j) {
            distance = findDistanceGPU(p, i, j);
            
            if (distance < NEIGHBORHOOD && p[j*4+3] == 0xff0000) {
                numI[i]++;
            }
        }
    }
}


void dumpResults(int index[]) {
    int i;
    FILE *fp;
    
    fp = fopen("./dump.out", "w");
    
    for (i=0; i<NUMPARTICLES; i++) {
        fprintf(fp, "%d %d\n", i, index[i]);
    }
    
    fclose(fp);
}
